/******************************************************************************
# SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
# SPDX-License-Identifier: Apache-2.0
#
# Licensed under the Apache License, Version 2.0 (the "License");
# you may not use this file except in compliance with the License.
# You may obtain a copy of the License at
#
# http://www.apache.org/licenses/LICENSE-2.0
#
# Unless required by applicable law or agreed to in writing, software
# distributed under the License is distributed on an "AS IS" BASIS,
# WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
# See the License for the specific language governing permissions and
# limitations under the License.
******************************************************************************/
// Copyright (c) 2024, Tri Dao.
// Copyright (c) 2024, NVIDIA CORPORATION & AFFILIATES.
// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_fwd_launch_template.h"

template<>
void run_mha_fwd_<90, cutlass::bfloat16_t, 256>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim256<90, cutlass::bfloat16_t>(params, stream);
}
